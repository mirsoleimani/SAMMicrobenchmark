#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <utility.h>
#include <hip/hip_runtime.h>
#include <>


#define DEFAULTMEMSIZE (1<<2)//4B

#define MINBLOCKSIZE 1
#define MAXBLOCKSIZE 1024
#define MAXGRIDSIZE 65535

#define NUMREPEAT 1000000
#define PINNED 0

int main()
{
    size_t memSize = DEFAULTMEMSIZE;
    size_t nWords = (memSize)/sizeof(float);

    //Initialize Host memory
    float *h_iData;
    float *h_oData;
    if(PINNED)
    {
        hipHostAlloc((void **)&h_iData,memSize,0);
        hipHostAlloc((void **)&h_oData,memSize,0);
    }
    else
    {
        h_iData = new float[nWords];
        h_oData = new float[nWords];
    }

    for(unsigned int i=0;i<nWords;i++)
    {
        h_oData[i]= (float) (i&(0xFF));
    }

    //initialize Device memory
    float *d_iData;
    float *d_oData;

    hipMalloc((void **)&d_iData,memSize);
    CUDA_HANDLE_ERROR();
    hipMalloc((void **)&d_oData,memSize);
    CUDA_HANDLE_ERROR();


    hipEvent_t start, stop;
    float time,latency;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    CUDA_HANDLE_ERROR();

    hipEventRecord(start,0);

        if(PINNED)
        {
		for(int i=0;i<NUMREPEAT;i++)
		{
            hipMemcpyAsync(d_iData,h_iData,memSize,hipMemcpyHostToDevice,0);
		}
        }
        else
        {
		for(int i=0;i<NUMREPEAT;i++)
		{
            hipMemcpy(d_iData, h_iData, memSize, hipMemcpyHostToDevice);
		}
        }
//        hipDeviceSynchronize();
    

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    time=0.0f;
    hipEventElapsedTime(&time,start,stop);

    //hipMemcpy(h_oData, d_oData, memSize, hipMemcpyDeviceToHost);
    //CUDA_HANDLE_ERROR();


    time /= 1.e3;
    latency = time/((float)NUMREPEAT);

    printf("memory copy host trasfer lunch overhead is:%0.15f\n",latency);

   if(PINNED)
   {
	   hipHostFree(d_iData);
	   hipHostFree(d_oData);
   }
   else
   {
 free(h_iData);
 free(h_oData);



   }
   
     hipFree(d_iData);  
    hipFree(d_oData);
}
