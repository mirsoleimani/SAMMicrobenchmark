#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <utility.h>


#define DEFAULTMEMSIZE (1<<2)//4B

#define MINBLOCKSIZE 1
#define MAXBLOCKSIZE 1024
#define MAXGRIDSIZE 65535

#define NUMREPEAT 1000000

int main()
{
    size_t memSize = DEFAULTMEMSIZE;
    size_t nWords = (memSize)/sizeof(float);
    
    //Initialize Host memory
    float *h_iData;
    float *h_oData;
    h_iData = new float[nWords];
    h_oData = new float[nWords];

    for(unsigned int i=0;i<nWords;i++)
    {
        h_iData[i]= (float) (i&(0xFF));
    }

    //initialize Device memory
    float *d_iData;
    float *d_oData;
    hipMalloc((void **)&d_iData,memSize);
    CUDA_HANDLE_ERROR();
    hipMalloc((void **)&d_oData,memSize);
    CUDA_HANDLE_ERROR();


    hipEvent_t start, stop;
    float time,latency;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    CUDA_HANDLE_ERROR();

    hipEventRecord(start,0);
    
    for(int i=0;i<NUMREPEAT;i++)
    {
        hipMemcpy(d_iData, h_iData, memSize, hipMemcpyHostToDevice);
	hipDeviceSynchronize();
    }

    hipEventRecord(stop,0);
   // hipEventSynchronize(stop);
    time=0.0f;
    hipEventElapsedTime(&time,start,stop);
    
    //hipMemcpy(h_oData, d_oData, memSize, hipMemcpyDeviceToHost);
    //CUDA_HANDLE_ERROR();


    time /= 1.e3;
    latency = time/((float)NUMREPEAT);

    printf("kernel lunch overhead is:%0.15f\n",latency);
}
