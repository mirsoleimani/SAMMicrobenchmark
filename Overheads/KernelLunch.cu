#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <utility.h>

#define DEFAULTMEMSIZE (1<<20)

#define MINBLOCKSIZE 1
#define MAXBLOCKSIZE 1024
#define MAXGRIDSIZE 65535

#define NUMREPEAT 100

__global__ void SimpleCopy(float *oData, float *iData)
{
    int xId = blockIdx.x * blockDim.x + threadIdx.x;
    oData[xId] = iData[xId];

}

int main()
{
    size_t memSize = DEFAULTMEMSIZE;
    size_t nWords = (memSize)/sizeof(float);
    
    //Initialize Host memory
    float *h_iData;
    float *h_oData;
    h_iData = new float[nWords];
    h_oData = new float[nWords];

    for(unsigned int i=0;i<nWords;i++)
    {
        h_iData[i]= (float) (i&(0xFF));
    }

    //initialize Device memory
    float *d_iData;
    float *d_oData;
    hipMalloc((void **)&d_iData,memSize);
    CUDA_HANDLE_ERROR();
    hipMalloc((void **)&d_oData,memSize);
    CUDA_HANDLE_ERROR();

    hipMemcpy(d_iData, h_iData, memSize, hipMemcpyHostToDevice);


    dim3 blockSize = 1;
    dim3 gridSize = 1;

    hipEvent_t start, stop;
    float time,latency;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    CUDA_HANDLE_ERROR();

    hipEventRecord(start);
    
    for(int i=0;i<NUMREPEAT;i++)
    {
        SimpleCopy<<<gridSize,blockSize>>>(h_oData,h_iData);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time,start,stop);

    latency = time/(1e3*(double)NUMREPEAT);

    printf("kernel lunch overhead is:%0.10f\n",latency);
}