#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <utility.h>

#define DEFAULTMEMSIZE (1<<20)

#define MINBLOCKSIZE 1
#define MAXBLOCKSIZE 1024
#define MAXGRIDSIZE 65535

#define NUMREPEAT 1000000

__global__ void SimpleCopy(float *oData, float *iData)
{
    int xId = blockIdx.x * blockDim.x + threadIdx.x;
    oData[xId] = iData[xId]+1000;
}

int main()
{
    size_t memSize = DEFAULTMEMSIZE;
    size_t nWords = (memSize)/sizeof(float);
    
    //Initialize Host memory
    float *h_iData;
    float *h_oData;
    h_iData = new float[nWords];
    h_oData = new float[nWords];

    for(unsigned int i=0;i<nWords;i++)
    {
        h_iData[i]= (float) (i&(0xFF));
    }

    //initialize Device memory
    float *d_iData;
    float *d_oData;
    hipMalloc((void **)&d_iData,memSize);
    CUDA_HANDLE_ERROR();
    hipMalloc((void **)&d_oData,memSize);
    CUDA_HANDLE_ERROR();

    hipMemcpy(d_iData, h_iData, memSize, hipMemcpyHostToDevice);


    dim3 blockSize(1,1,1);
    dim3 gridSize(1,1,1);

    hipEvent_t start, stop;
    float time,latency;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    CUDA_HANDLE_ERROR();

    hipEventRecord(start,0);
    
    for(int i=0;i<NUMREPEAT;i++)
    {
        SimpleCopy<<<gridSize,blockSize>>>(h_oData,h_iData);
        hipDeviceSynchronize();
        hipMemcpy(h_oData, d_oData, memSize, hipMemcpyDeviceToHost);
    }

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    time=0.0f;
    hipEventElapsedTime(&time,start,stop);

    
    time /= 1.e3;
    latency = time/((float)NUMREPEAT);

    printf("kernel lunch overhead is:%0.5f\n",time);
}
