#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <repeat.h>
#include <utility.h>
#include <hip/hip_runtime.h>
#include <>

#define DEFAULTMEMSIZE (1<<20)
#define DEFAULTSTEPSIZE (1<<20)

#define MINBLOCKSIZE 1
#define MAXBLOCKSIZE 1024
#define MAXGRIDSIZE 65535

#define CLOCK 0.000000000713

float *h_iData;
float *h_oData;
float *d_iData;
float *d_oData;

void PrintResult(char *fName,unsigned int *size,unsigned int *stride,
    float *latency, unsigned int *clock,unsigned int count)
{
    FILE *fOut;
    fOut = fopen(fName,"w");

    fprintf(fOut,"#ArraySize(B)\tStride(B)\tlatency(ns)\tclock\n");
    for(int i=0;i<count;i++)
    {
        fprintf(fOut,"%d\t%d\t%d\t%d\n",size[i],stride[i],latency[i],clock[i]);
    }
    fclose(fOut);
}
//***SimpleCopy_Start***
__global__ void SimpleCopy(float *oData, float *iData)
{
    int xId = blockIdx.x * blockDim.x + threadIdx.x;
    oData[xId] = iData[xId];

}
//***SimpleCopy_End***

//***StrideCopy_Start***
__global__ void StrideCopy(float *oData, float *iData,int stride)
{
    int xId = (blockIdx.x*blockDim.x+threadIdx.x)*stride;
    oData[xId] = iData[xId];
}
//***StrideCopy_End***

//***StrideAccess_Start***
__global__ void StrideAccess(float *oData, float *iData,int nWords)
{
    unsigned int xId=0;

#pragma unroll 512
    for(int i=0;i<nWords;i++)
    {
        xId= iData[xId];
    }
    
    oData[0]=iData[xId];	
}
//***StrideAccess_End***

void RunStrideAccess(int stride,int nWords)
{
    //***RunStideAccessFill_Start***
    for(unsigned int i=0;i<nWords;i++)
    {
        h_oData[i]= (float)((i+stride)%nWords);
    }
    //***RunStrideAccessFill_End***

    hipMemcpy(d_iData, h_oData, nWords*sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = nWords/blockSize;

    hipEvent_t start, stop;
    float time,latency;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    CUDA_HANDLE_ERROR();

    hipEventRecord(start,0);

    StrideAccess<<<1,1>>>(d_oData,d_iData,nWords);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    CUDA_HANDLE_ERROR();

    hipDeviceSynchronize();    

    hipMemcpy(h_iData, d_oData, nWords*sizeof(float), hipMemcpyDeviceToHost);

    time=0.0f;
    hipEventElapsedTime(&time,start,stop);

    time /= 1.e3;
    latency = (time*1.0)/(float)nWords;
    int clocks = (int) latency/CLOCK;


    printf("data:%f, time:%f, stride:%d, latency:%0.10f, clocks:%d\n",h_iData[0],time,stride,latency,clocks);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    CUDA_HANDLE_ERROR();
}
void TestLatency(size_t memSize)
{
    size_t nWords = (memSize)/sizeof(float);

    //Initialize Host memory
    h_iData = new float[nWords];
    h_oData = new float[nWords];

    //initialize Device memory
    hipMalloc((void **)&d_iData,memSize);
    CUDA_HANDLE_ERROR();
    hipMalloc((void **)&d_oData,memSize);
    CUDA_HANDLE_ERROR();


    for(int stride=1;stride <= nWords/2; stride*=2)
    {
        RunStrideAccess(stride, nWords);
    }

    hipFree(d_iData);
    hipFree(d_oData);

    free(h_iData);
    free(h_oData);

}

void TestBandwidth(size_t memSize)
{
    
    size_t nWords = (memSize)/sizeof(float);

    //Initialize Host memory
    h_iData = new float[nWords];
    h_oData = new float[nWords];

    //initialize Device memory
    hipMalloc((void **)&d_iData,memSize);
    CUDA_HANDLE_ERROR();
    hipMalloc((void **)&d_oData,memSize);
    CUDA_HANDLE_ERROR();

    hipFree(d_iData);
    hipFree(d_oData);

    free(h_iData);
    free(h_oData);

}

int main()
{
    size_t memSize = 128*DEFAULTMEMSIZE;
    for(size_t memSize=DEFAULTMEMSIZE;memSize<256*DEFAULTMEMSIZE;memSize+=DEFAULTSTEPSIZE)
    {
        TestLatency(memSize);
    }

    return 0;
}
