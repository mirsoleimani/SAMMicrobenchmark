#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <repeat.h>
#include <utility.h>
#include <hip/hip_runtime.h>
#include <>


#define DEFAULTMEMSIZE (1<<10)//1KB

#define MINBLOCKSIZE 1
#define MAXBLOCKSIZE 1024
#define MAXGRIDSIZE 65535

#define CLOCK 0.000000000713

unsigned int *h_iData;
unsigned int *h_oData;
unsigned int *d_iData;
unsigned int *d_oData;

void PrintResult(char *fName,unsigned int *size,unsigned int *stride,
    unsigned int *latency, unsigned int *clock,unsigned int count)
{
    FILE *fOut;
    fOut = fopen(fName,"w");

    fprintf(fOut,"#ArraySize(B)\tStride(B)\tlatency(ns)\tclock\n");
    for(int i=0;i<count;i++)
    {
        fprintf(fOut,"%d\t%d\t%d\t%d\n",size[i],stride[i],latency[i],clock[i]);
    }
    fclose(fOut);
}
//***SimpleCopy_Start***
__global__ void SimpleCopy(float *oData, float *iData)
{
    int xId = blockIdx.x * blockDim.x + threadIdx.x;
    oData[xId] = iData[xId];

}
//***SimpleCopy_End***

//***StrideCopy_Start***
__global__ void StrideCopy(float *oData, float *iData,int stride)
{
    int xId = (blockIdx.x*blockDim.x+threadIdx.x)*stride;
    oData[xId] = iData[xId];
}
//***StrideCopy_End***

//***StrideAccess_Start***
extern "C"
 __global__ void StrideAccess(unsigned int *oData, unsigned int *iData, int itr)
{
    unsigned int xId=0;
    unsigned int start=0,stop=0;
    volatile unsigned int sumTime=0;


    for(int i=0;i<itr;i++)
    {
       // start = clock();
        repeat512(xId= iData[xId];)//dependency
       // stop = clock();

       // sumTime += stop-start;
    }

    oData[0]=iData[xId];
    oData[1]=sumTime;
}
//***StrideAccess_End***



void RunStrideAccess(int stride,int nWords, int itr)
{
    //***RunStrideAccessFill_Start***
    //Lay out path of memory references in array
    for(unsigned int i=0;i<nWords;i++)
    {
        h_oData[i]= ((i+stride)%nWords);
    }
    //***RunStrideAccessFill_End***

    hipMemcpy(d_iData, h_oData, (nWords+1)*sizeof(unsigned int), hipMemcpyHostToDevice);

    dim3 blockSize = dim3(1,1);
    dim3 gridSize = dim3(1,1,1);

    hipEvent_t start, stop;
    float time,latency;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    CUDA_HANDLE_ERROR();

    
    hipEventRecord(start,0);

    StrideAccess<<<1,1>>>(d_oData,d_iData,itr);

    hipEventRecord(stop,0);
    hipDeviceSynchronize();    
    hipEventSynchronize(stop);
    CUDA_HANDLE_ERROR();

    hipMemcpy(h_iData, d_oData, nWords*sizeof(unsigned int), hipMemcpyDeviceToHost);

    time=0.0f;
    hipEventElapsedTime(&time,start,stop);

    time /= 1.e3;
    latency = (time*1.0)/(float)(itr*512);
    unsigned int clocks = (latency/CLOCK);

    latency*=1.e9;
    int c = (float)h_iData[1]/(float)((itr)*512);
    printf("%d\t%f\t%d\t%0.0f\t%d\t%d\n",nWords*sizeof(int),time,stride*sizeof(int),latency,clocks,c);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    CUDA_HANDLE_ERROR();
}
void TestLatency(size_t memSize)
{
    size_t nWords = (memSize)/sizeof(unsigned int);
    int itr=200;

    //Initialize Host memory
    h_iData = new unsigned int[nWords+1];
    h_oData = new unsigned int[nWords+1];

    //initialize Device memory
    hipMalloc((void **)&d_iData,(nWords+1)*sizeof(int));
    CUDA_HANDLE_ERROR();
    hipMalloc((void **)&d_oData,(nWords+1)*sizeof(int));
    CUDA_HANDLE_ERROR();

    printf("#size\ttime\tstride\tlatency(ns)\tclocks\n");
    for(int stride=1;stride <= nWords/2; stride*=2)
    {
        RunStrideAccess(stride, nWords,itr);
    }

    hipFree(d_iData);
    hipFree(d_oData);

    free(h_iData);
    free(h_oData);

}

void TestBandwidth(size_t memSize)
{
    
    size_t nWords = (memSize)/sizeof(unsigned int);

    //Initialize Host memory
    h_iData = new unsigned int[nWords];
    h_oData = new unsigned int[nWords];

    //initialize Device memory
    hipMalloc((void **)&d_iData,memSize);
    CUDA_HANDLE_ERROR();
    hipMalloc((void **)&d_oData,memSize);
    CUDA_HANDLE_ERROR();

    hipFree(d_iData);
    hipFree(d_oData);

    free(h_iData);
    free(h_oData);

}

int main()
{
	hipDeviceSetCacheConfig(hipFuncCachePreferL1);
    for(size_t memSize=4*DEFAULTMEMSIZE;memSize<=512*1024*DEFAULTMEMSIZE;memSize*=2)
    {
        TestLatency(memSize);
    }

    return 0;
}
