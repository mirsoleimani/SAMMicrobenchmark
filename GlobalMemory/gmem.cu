#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <repeat.h>
#include <utility.h>
#include <hip/hip_runtime.h>
#include <>

#define DEFAULTMEMSIZE (1<<20)
#define DEFAULTSTEPSIZE (1<<20)

#define MINBLOCKSIZE 1
#define MAXBLOCKSIZE 1024
#define MAXGRIDSIZE 65535

float *h_iData;
float *h_oData;
float *d_iData;
float *d_oData;

//***SimpleCopy_Start***
__global__ void SimpleCopy(float *oData, float *iData)
{
    int xId = blockIdx.x * blockDim.x + threadIdx.x;
    oData[xId] = iData[xId];

}
//***SimpleCopy_End***

//***StrideCopy_Start***
__global__ void StrideCopy(float *oData, float *iData,int stride)
{
    int xId = (blockIdx.x*blockDim.x+threadIdx.x)*stride;
    oData[xId] = iData[xId];
}
//***StrideCopy_End***

//***StrideAccess_Start***
__global__ void StrideAccess(float *oData, float *iData,int nWords)
{
    unsigned int xId=0;
    float sum=0;

#pragma unroll 1024
    for(int i=0;i<nWords;i++)
    {
        xId= iData[xId];
    }

    oData[0]=xId;	
}
//***StrideAccess_End***

void RunStrideAccess(int stride,int nWords, int memSize,int nRepeats)
{
    for(unsigned int i=0;i<nWords;i++)
    {
        h_oData[i]= (float)((i+stride)%nWords);
    }

    hipMemcpy(d_iData, h_oData, memSize, hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = nWords/blockSize;

    hipEvent_t start, stop;
    float time,latency;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    CUDA_HANDLE_ERROR();

    hipEventRecord(start,0);

    //  for(int i=0;i<nRepeats;i++)
    //{
    StrideAccess<<<1,1>>>(d_oData,d_iData,nWords);
    // }

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    hipMemcpy(h_iData, d_oData, memSize, hipMemcpyDeviceToHost);

    time=0.0f;
    hipEventElapsedTime(&time,start,stop);

    time /= 1.e3;
    latency = (time*1.0)/(float)nWords;
    float clock = latency/7.e-11;


    printf("data:%f, time:%f, stride:%d, latency:%0.10f, clock:%f\n",h_iData[0],time,stride,latency,clock);


}
void TestLatency()
{
    size_t memSize = 128*DEFAULTMEMSIZE;
    size_t nWords = (memSize)/sizeof(float);
    int nRepeats=20;

    //Initialize Host memory
    h_iData = new float[nWords];
    h_oData = new float[nWords];

    //initialize Device memory
    hipMalloc((void **)&d_iData,memSize);
    CUDA_HANDLE_ERROR();
    hipMalloc((void **)&d_oData,memSize);
    CUDA_HANDLE_ERROR();
    int stride=2;
    //    for(int stride=1;stride <= nWords/2; stride*=2)
    //    {
    RunStrideAccess(stride, nWords, memSize, nRepeats);
    //    }

    hipFree(d_iData);
    hipFree(d_oData);

    free(h_iData);
    free(h_oData);

}

void TestBandwidth()
{
    size_t memSize = 128*DEFAULTMEMSIZE;
    size_t nWords = (memSize)/sizeof(float);

    //Initialize Host memory
    h_iData = new float[nWords];
    h_oData = new float[nWords];

    //initialize Device memory
    hipMalloc((void **)&d_iData,memSize);
    CUDA_HANDLE_ERROR();
    hipMalloc((void **)&d_oData,memSize);
    CUDA_HANDLE_ERROR();

    hipFree(d_iData);
    hipFree(d_oData);

    free(h_iData);
    free(h_oData);
}

int main()
{
    TestLatency();


    return 0;
}
