#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <repeat.h>
#include <utility.h>
#include <hip/hip_runtime.h>
#include <>

#define DEFAULTMEMSIZE (1<<20)
#define DEFAULTSTEPSIZE (1<<20)

#define MINBLOCKSIZE 1
#define MAXBLOCKSIZE 1024
#define MAXGRIDSIZE 65535

#define CLOCK 0.000000000713

float *h_iData;
float *h_oData;
float *d_iData;
float *d_oData;

//***SimpleCopy_Start***
__global__ void SimpleCopy(float *oData, float *iData)
{
    int xId = blockIdx.x * blockDim.x + threadIdx.x;
    oData[xId] = iData[xId];

}
//***SimpleCopy_End***

//***StrideCopy_Start***
__global__ void StrideCopy(float *oData, float *iData,int stride)
{
    int xId = (blockIdx.x*blockDim.x+threadIdx.x)*stride;
    oData[xId] = iData[xId];
}
//***StrideCopy_End***

//***StrideAccess_Start***
__global__ void StrideAccess(float *oData, float *iData,int nWords)
{
    unsigned int xId=0;
    float sum=0;

#pragma unroll 512
    for(int i=0;i<nWords;i++)
    {
        xId= iData[xId];
    }

    oData[0]=xId;	
}
//***StrideAccess_End***

void RunStrideAccess(int stride,int nWords)
{
    for(unsigned int i=0;i<nWords;i++)
    {
        h_oData[i]= (float)((i+stride)%nWords);
    }

    hipMemcpy(d_iData, h_oData, nWords*sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = nWords/blockSize;

    hipEvent_t start, stop;
    float time,latency;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    CUDA_HANDLE_ERROR();

    hipEventRecord(start,0);

    StrideAccess<<<1,1>>>(d_oData,d_iData,nWords);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    hipMemcpy(h_iData, d_oData, nWords*sizeof(float), hipMemcpyDeviceToHost);

    time=0.0f;
    hipEventElapsedTime(&time,start,stop);

    time /= 1.e3;
    latency = (time*1.0)/(float)nWords;
    float clocks = latency/CLOCK;


    printf("data:%f, time:%f, stride:%d, latency:%0.10f, clocks:%f\n",h_iData[0],time,stride,latency,clocks);

    hipEventDestroy(start);
    hipEventDestroy(stop);
}
void TestLatency()
{
    size_t memSize = 128*DEFAULTMEMSIZE;
    size_t nWords = (memSize)/sizeof(float);
    int nRepeats=20;

    //Initialize Host memory
    h_iData = new float[nWords];
    h_oData = new float[nWords];

    //initialize Device memory
    hipMalloc((void **)&d_iData,memSize);
    CUDA_HANDLE_ERROR();
    hipMalloc((void **)&d_oData,memSize);
    CUDA_HANDLE_ERROR();


    for(int stride=1;stride <= nWords/2; stride*=2)
    {
        RunStrideAccess(stride, nWords);
    }

    hipFree(d_iData);
    hipFree(d_oData);

    free(h_iData);
    free(h_oData);

}

void TestBandwidth()
{
    size_t memSize = 128*DEFAULTMEMSIZE;
    size_t nWords = (memSize)/sizeof(float);

    //Initialize Host memory
    h_iData = new float[nWords];
    h_oData = new float[nWords];

    //initialize Device memory
    hipMalloc((void **)&d_iData,memSize);
    CUDA_HANDLE_ERROR();
    hipMalloc((void **)&d_oData,memSize);
    CUDA_HANDLE_ERROR();

    hipFree(d_iData);
    hipFree(d_oData);

    free(h_iData);
    free(h_oData);

}

int main()
{
    TestLatency();


    return 0;
}
