#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <repeat.h>
#include <utility.h>
#include <hip/hip_runtime.h>
#include <>


#define DEFAULTMEMSIZE (1<<10)//1KB

#define MINBLOCKSIZE 1
#define MAXBLOCKSIZE 1024
#define MAXGRIDSIZE 65535

#define CLOCK 0.000000000713

#define LOOP 512

unsigned int *h_iData;
unsigned int *h_oData;
unsigned int *d_iData;
unsigned int *d_oData;

void PrintResult(char *fName,unsigned int *size,unsigned int *stride,
    unsigned int *latency, unsigned int *clock,unsigned int count)
{
    FILE *fOut;
    fOut = fopen(fName,"w");

    fprintf(fOut,"#ArraySize(B)\tStride(B)\tlatency(ns)\tclock\n");
    for(int i=0;i<count;i++)
    {
        fprintf(fOut,"%d\t%d\t%d\t%d\n",size[i],stride[i],latency[i],clock[i]);
    }
    fclose(fOut);
}
//***SimpleCopy_Start***
__global__ void SimpleCopy(float *oData, float *iData)
{
    int xId = blockIdx.x * blockDim.x + threadIdx.x;
    oData[xId] = iData[xId];

}
//***SimpleCopy_End***

//***StrideCopy_Start***
__global__ void StrideCopy(float *oData, float *iData,int stride)
{
    int xId = (blockIdx.x*blockDim.x+threadIdx.x)*stride;
    oData[xId] = iData[xId];
}
//***StrideCopy_End***

//***StrideAccess_Start***
extern "C"
 __global__ void StrideAccess(unsigned int *oData, unsigned int *iData, int itr)
{
    unsigned int xId=0;
    unsigned int start=0,stop=0;
    unsigned int sumTime=0;

    for(int i=0;i<itr;i++)
    {
       start = clock();
       repeat512(xId= iData[xId];)//dependency
       stop = clock();

        sumTime += stop-start;
    }

    oData[0]=iData[xId];
    oData[1]=sumTime;
}
//***StrideAccess_End***



void RunStrideAccess(int stride,int nWords, int itr)
{
    //***RunStrideAccessFill_Start***
    //Lay out path of memory references in array
    for(unsigned int i=0;i<nWords;i++)
    {
        h_oData[i]= ((i+stride)%nWords);
    }
    //***RunStrideAccessFill_End***

    hipMemcpy(d_iData, h_oData, (nWords+1)*sizeof(unsigned int), hipMemcpyHostToDevice);

    dim3 blockSize = dim3(1,1);
    dim3 gridSize = dim3(1,1,1);

    hipEvent_t start, stop;
    float time,latency;
    unsigned int clocks;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    CUDA_HANDLE_ERROR();

    
    hipEventRecord(start,0);

    StrideAccess<<<1,1>>>(d_oData,d_iData,itr);

    hipEventRecord(stop,0);
    hipDeviceSynchronize();    
    hipEventSynchronize(stop);
    CUDA_HANDLE_ERROR();

    hipMemcpy(h_iData, d_oData, nWords*sizeof(unsigned int), hipMemcpyDeviceToHost);

    time=0.0f;
    clocks=0;
    hipEventElapsedTime(&time,start,stop);

    time /= 1.e3;
    //latency = (time*1.0)/(float)(itr*512);
    //clocks = (latency/CLOCK);
    //latency*=1.e9;

    clocks = (float)h_iData[1]/(float)((itr)*512);
	latency = clocks*CLOCK;
	latency *=1.e9;

    printf("%d,%0.2f,%d,%0.0f,%d\n",nWords*sizeof(int),time,stride*sizeof(int),latency,clocks);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    CUDA_HANDLE_ERROR();
}
void TestLatency(size_t memSize)
{
    size_t nWords = (memSize)/sizeof(unsigned int);
    int itr=200;

    //Initialize Host memory
    h_iData = new unsigned int[nWords+1];
    h_oData = new unsigned int[nWords+1];

    //initialize Device memory
    hipMalloc((void **)&d_iData,(nWords+1)*sizeof(int));
    CUDA_HANDLE_ERROR();
    hipMalloc((void **)&d_oData,(nWords+1)*sizeof(int));
    CUDA_HANDLE_ERROR();

    printf("#size(b),time(s),stride(b),latency(ns),clocks\n");
    for(int stride=1;stride <= nWords; stride*=2)
    {
        RunStrideAccess(stride, nWords,itr);
    }

    hipFree(d_iData);
    hipFree(d_oData);

    free(h_iData);
    free(h_oData);

}

void TestBandwidth(size_t memSize)
{
    
    size_t nWords = (memSize)/sizeof(unsigned int);

    //Initialize Host memory
    h_iData = new unsigned int[nWords];
    h_oData = new unsigned int[nWords];

    //initialize Device memory
    hipMalloc((void **)&d_iData,memSize);
    CUDA_HANDLE_ERROR();
    hipMalloc((void **)&d_oData,memSize);
    CUDA_HANDLE_ERROR();

    hipFree(d_iData);
    hipFree(d_oData);

    free(h_iData);
    free(h_oData);

}

int main()
{
	hipDeviceSetCacheConfig(hipFuncCachePreferL1);
    for(size_t memSize=4*DEFAULTMEMSIZE;memSize<=8*1024*DEFAULTMEMSIZE;memSize+=DEFAULTMEMSIZE)
    {
        TestLatency(memSize);
    }

    return 0;
}
