#include <stdio.h>
#include <hip/hip_runtime.h>

#define DEFAULTMEMSIZE (1<<20)

#define MINBLOCKSIZE 1
#define MAXBLOCKSIZE 1024
#define MAXGRIDSIZE 65535


__global__ void SimpleCopy(float *oData, float *iData)
{
    int xId = blockIdx.x * blockDim.x + threadIdx.x;
    oData[xId] = iData[xId]++;
}


int main()
{
    size_t memSize = 32*DEFAULTMEMSIZE;
    size_t nWords = (memSize)/sizeof(float);
    
    //Initialize Host memory
    float *h_iData;
    float *h_oData;
    h_iData = new float[nWords];
    h_oData = new float[nWords];

    for(unsigned int i=0;i<nWords;i++)
    {
        h_iData[i]= (float) (i&(0xFF));
    }

    //initialize Device memory
    float *d_iData;
    float *d_oData;
    hipMalloc((void **)&d_iData,memSize);
    //CUDA_HANDLE_ERROR();
    hipMalloc((void **)&d_oData,memSize);
    //CUDA_HANDLE_ERROR();

    hipMemcpy(d_iData, h_iData, memSize, hipMemcpyHostToDevice);


    int blockSize = 256;
    int gridSize = nWords/blockSize;

    //cudaEvent_t start, stop;

}
