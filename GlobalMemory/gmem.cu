#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <repeat.h>
#include <utility.h>
#include <hip/hip_runtime.h>
#include <>

#define DEFAULTMEMSIZE (1<<20)
#define DEFAULTSTEPSIZE (1<<20)

#define MINBLOCKSIZE 1
#define MAXBLOCKSIZE 1024
#define MAXGRIDSIZE 65535

#define CLOCK 0.000000000713

char *h_iData;
char *h_oData;
char *d_iData;
char *d_oData;

void PrintResult(char *fName,unsigned int *size,unsigned int *stride,
    unsigned int *latency, unsigned int *clock,unsigned int count)
{
    FILE *fOut;
    fOut = fopen(fName,"w");

    fprintf(fOut,"#ArraySize(B)\tStride(B)\tlatency(ns)\tclock\n");
    for(int i=0;i<count;i++)
    {
        fprintf(fOut,"%d\t%d\t%d\t%d\n",size[i],stride[i],latency[i],clock[i]);
    }
    fclose(fOut);
}
//***SimpleCopy_Start***
__global__ void SimpleCopy(float *oData, float *iData)
{
    int xId = blockIdx.x * blockDim.x + threadIdx.x;
    oData[xId] = iData[xId];

}
//***SimpleCopy_End***

//***StrideCopy_Start***
__global__ void StrideCopy(float *oData, float *iData,int stride)
{
    int xId = (blockIdx.x*blockDim.x+threadIdx.x)*stride;
    oData[xId] = iData[xId];
}
//***StrideCopy_End***

//***StrideAccess_Start***
__global__ void StrideAccess(char *oData, char *iData,int nWords)
{
    unsigned int xId=0;
    unsigned int start,stop;

#pragma unroll 2048
    for(int i=0;i<nWords;i++)
    {
        xId= iData[xId];
    }

    oData[0]=iData[xId];
}
//***StrideAccess_End***

void RunStrideAccess(int stride,int nWords)
{
    //***RunStideAccessFill_Start***
    for(unsigned int i=0;i<nWords;i++)
    {
        h_oData[i]= (char)((i+stride)%nWords);
    }
    //***RunStrideAccessFill_End***

    hipMemcpy(d_iData, h_oData, nWords*sizeof(char), hipMemcpyHostToDevice);

    dim3 blockSize = dim3(1,1);
    dim3 gridSize = dim3(1,1,1);

    hipEvent_t start, stop;
    float time,latency;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    CUDA_HANDLE_ERROR();

    
    hipEventRecord(start,0);

    StrideAccess<<<1,1>>>(d_oData,d_iData,nWords);

    hipEventRecord(stop,0);
    hipDeviceSynchronize();    
    //hipEventSynchronize(stop);
    CUDA_HANDLE_ERROR();

    hipMemcpy(h_iData, d_oData, nWords*sizeof(char), hipMemcpyDeviceToHost);

    time=0.0f;
    hipEventElapsedTime(&time,start,stop);

    time /= 1.e3;
    latency = (time*1.0)/(float)nWords;
    unsigned int clocks = (latency/CLOCK);

    latency*=1.e9;
    
    printf("%d\t%f\t%d\t%0.0f\t%d\n",nWords*sizeof(char),time,stride,latency,clocks);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    CUDA_HANDLE_ERROR();
}
void TestLatency(size_t memSize)
{
    size_t nWords = (memSize)/sizeof(char);

    //Initialize Host memory
    h_iData = new char[nWords];
    h_oData = new char[nWords];

    //initialize Device memory
    hipMalloc((void **)&d_iData,memSize);
    CUDA_HANDLE_ERROR();
    hipMalloc((void **)&d_oData,memSize);
    CUDA_HANDLE_ERROR();

    printf("#size\ttime\tstride\tlatency(ns)\tclocks\n");
    for(int stride=1;stride <= nWords/2; stride*=2)
    {
        RunStrideAccess(stride, nWords);
    }

    hipFree(d_iData);
    hipFree(d_oData);

    free(h_iData);
    free(h_oData);

}

void TestBandwidth(size_t memSize)
{
    
    size_t nWords = (memSize)/sizeof(char);

    //Initialize Host memory
    h_iData = new char[nWords];
    h_oData = new char[nWords];

    //initialize Device memory
    hipMalloc((void **)&d_iData,memSize);
    CUDA_HANDLE_ERROR();
    hipMalloc((void **)&d_oData,memSize);
    CUDA_HANDLE_ERROR();

    hipFree(d_iData);
    hipFree(d_oData);

    free(h_iData);
    free(h_oData);

}

int main()
{
    for(size_t memSize=2*1024;memSize<=16*DEFAULTMEMSIZE;memSize*=2)
    {
        TestLatency(memSize);
    }

    return 0;
}
