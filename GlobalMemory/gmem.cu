#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <repeat.h>
#include <utility.h>
#include <hip/hip_runtime.h>
#include <>


#define DEFAULTMEMSIZE (1<<10)//1KB

#define MINBLOCKSIZE 1
#define MAXBLOCKSIZE 1024
#define MAXGRIDSIZE 65535

#define CLOCK 0.000000000713

unsigned int *h_iData;
unsigned int *h_oData;
unsigned int *d_iData;
unsigned int *d_oData;

void PrintResult(char *fName,unsigned int *size,unsigned int *stride,
    unsigned int *latency, unsigned int *clock,unsigned int count)
{
    FILE *fOut;
    fOut = fopen(fName,"w");

    fprintf(fOut,"#ArraySize(B)\tStride(B)\tlatency(ns)\tclock\n");
    for(int i=0;i<count;i++)
    {
        fprintf(fOut,"%d\t%d\t%d\t%d\n",size[i],stride[i],latency[i],clock[i]);
    }
    fclose(fOut);
}
//***SimpleCopy_Start***
__global__ void SimpleCopy(float *oData, float *iData)
{
    int xId = blockIdx.x * blockDim.x + threadIdx.x;
    oData[xId] = iData[xId];

}
//***SimpleCopy_End***

//***StrideCopy_Start***
__global__ void StrideCopy(float *oData, float *iData,int stride)
{
    int xId = (blockIdx.x*blockDim.x+threadIdx.x)*stride;
    oData[xId] = iData[xId];
}
//***StrideCopy_End***

//***StrideAccess_Start***
__global__ void StrideAccess(unsigned int *oData, unsigned int *iData, int itr)
{
    unsigned int xId=0;
    unsigned int start=0,stop=0;
    volatile unsigned int sumTime=0;


//#pragma unroll 256
    for(int i=0;i<itr;i++)
    {
        start = clock();
        repeat256(xId= iData[xId];)//dependency
        stop = clock();

        sumtime += stop-start;
    }

    oData[0]=iData[xId];
    oData[1]=sumTime;
}
//***StrideAccess_End***



void RunStrideAccess(int stride,int nWords, int itr)
{
    //***RunStrideAccessFill_Start***
    //Lay out path of memory references in array
    for(unsigned int i=0;i<nWords;i++)
    {
        h_oData[i]= ((i+stride)%nWords);
    }
    //***RunStrideAccessFill_End***

    hipMemcpy(d_iData, h_oData, (nWords+1)*sizeof(unsigned int), hipMemcpyHostToDevice);

    dim3 blockSize = dim3(1,1);
    dim3 gridSize = dim3(1,1,1);

    hipEvent_t start, stop;
    float time,latency;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    CUDA_HANDLE_ERROR();

    
    hipEventRecord(start,0);

    StrideAccess<<<1,1>>>(d_oData,d_iData,itr);

    hipEventRecord(stop,0);
    hipDeviceSynchronize();    
    hipEventSynchronize(stop);
    CUDA_HANDLE_ERROR();

    hipMemcpy(h_iData, d_oData, nWords*sizeof(unsigned int), hipMemcpyDeviceToHost);

    time=0.0f;
    hipEventElapsedTime(&time,start,stop);

    time /= 1.e3;
    latency = (time*1.0)/(float)(itr*256);
    unsigned int clocks = (latency/CLOCK);

    latency*=1.e9;
    
    printf("%d\t%f\t%d\t%0.0f\t%d\t%d\n",nWords*sizeof(int),time,stride*sizeof(int),latency,clocks,h_iData[1]);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    CUDA_HANDLE_ERROR();
}
void TestLatency(size_t memSize)
{
    size_t nWords = (memSize)/sizeof(unsigned int);
    int itr=100;

    //Initialize Host memory
    h_iData = new unsigned int[nWords];
    h_oData = new unsigned int[nWords+1];

    //initialize Device memory
    hipMalloc((void **)&d_iData,memSize);
    CUDA_HANDLE_ERROR();
    hipMalloc((void **)&d_oData,memSize);
    CUDA_HANDLE_ERROR();

    printf("#size\ttime\tstride\tlatency(ns)\tclocks\n");
    for(int stride=1;stride <= nWords/2; stride*=2)
    {
        RunStrideAccess(stride, nWords,itr);
    }

    hipFree(d_iData);
    hipFree(d_oData);

    free(h_iData);
    free(h_oData);

}

void TestBandwidth(size_t memSize)
{
    
    size_t nWords = (memSize)/sizeof(unsigned int);

    //Initialize Host memory
    h_iData = new unsigned int[nWords];
    h_oData = new unsigned int[nWords];

    //initialize Device memory
    hipMalloc((void **)&d_iData,memSize);
    CUDA_HANDLE_ERROR();
    hipMalloc((void **)&d_oData,memSize);
    CUDA_HANDLE_ERROR();

    hipFree(d_iData);
    hipFree(d_oData);

    free(h_iData);
    free(h_oData);

}

int main()
{
	hipDeviceSetCacheConfig(hipFuncCachePreferL1);
    for(size_t memSize=4*DEFAULTMEMSIZE;memSize<=8*1024*DEFAULTMEMSIZE;memSize*=2)
    {
        TestLatency(memSize);
    }

    return 0;
}
