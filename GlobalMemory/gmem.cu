#include <stdio.h>
#include <hip/hip_runtime.h>



#define DEFAULTMEMSIZE (1<<20)

#define MINBLOCKSIZE 1
#define MAXBLOCKSIZE 1024
#define MAXGRIDSIZE 65535

//***SimpleCopy_Start***
__global__ void SimpleCopy(float *oData, float *iData)
{
    int xId = blockIdx.x * blockDim.x + threadIdx.x;
    oData[xId] = iData[xId];

}
//***SimpleCopy_End***

//***StrideCopy_Start***
__global__ void StrideCopy(float *oData, float *iData,int stride)
{
    int xId = (blockIdx.x*blockDim.x+threadIdx.x)*stride;
    oData[xId] = iData[xId];
}
//***StrideCopy_End***

//***StrideAccess_Start***
__global__ void StrideAccess(float *oData, float *iData)
{

}
//***StrideAccess_End***

void TestLatency()
{

}

void TestBandwidth()
{

}

int main()
{
    size_t memSize = 32*DEFAULTMEMSIZE;
    size_t nWords = (memSize)/sizeof(float);
    
    //Initialize Host memory
    float *h_iData;
    float *h_oData;
    h_iData = new float[nWords];
    h_oData = new float[nWords];

    for(unsigned int i=0;i<nWords;i++)
    {
        h_iData[i]= (float) (i&(0xFF));
    }

    //initialize Device memory
    float *d_iData;
    float *d_oData;
    hipMalloc((void **)&d_iData,memSize);
    //CUDA_HANDLE_ERROR();
    hipMalloc((void **)&d_oData,memSize);
    //CUDA_HANDLE_ERROR();

    hipMemcpy(d_iData, h_iData, memSize, hipMemcpyHostToDevice);


    int blockSize = 256;
    int gridSize = nWords/blockSize;

    //cudaEvent_t start, stop;

    return 0;
}
